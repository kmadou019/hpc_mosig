
#include <hip/hip_runtime.h>
#include <iostream>

#define N 8
#define THREADS_PER_BLOCK 8

__global__ void prescan(float *g_idata, int n)
{
  extern __shared__ float temp[];
  // allocated on invocation
  int thid = threadIdx.x;
  int offset = 1;

  temp[thid] = g_idata[thid]; // load input into shared memory

  for (int d = n >> 1; d > 0; d = d >> 1)
  // build sum in place up the tree
  {
    __syncthreads();
    if (thid < d)
    {

      int ai = offset * (2 * thid + 1) - 1;
      int bi = offset * (2 * thid + 2) - 1;

      temp[bi] += temp[ai];
    }
    offset *= 2;
  }
  if (thid == 0)
  {
    temp[n - 1] = 0;
  } // clear the last element

  for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
  {
    offset = offset >> 1;
    __syncthreads();
    if (thid < d)
    {

      int ai = offset * (2 * thid + 1) - 1;
      int bi = offset * (2 * thid + 2) - 1;
      float t = temp[ai];
      temp[ai] = temp[bi];
      temp[bi] += t;
    }
  }
  __syncthreads();

  g_idata[thid] = temp[thid]; // write results to device memory
}

int main(){
    //int N = 10;
    float tab_host[N] = {0,1,2,3,4,5,6,7};
    float *tab_device;

    const float size = N*sizeof(float);
  
    // alloc mem on GPU
    hipMalloc( (void**)&tab_device, size );

    hipMemcpy(tab_device, tab_host, size, hipMemcpyHostToDevice);

    //const int blocksize = 8; 

    //dim3 dimBlock(blocksize , 1);
    //dim3 dimGrid(2,1);

    prescan<<<2,8,N * sizeof(float)>>>(tab_device, N);

    hipDeviceSynchronize();

    hipMemcpy(tab_host, tab_device, size, hipMemcpyDeviceToHost);
      
    for (size_t i = 0; i < N; i++)
    {
        std::cout<<tab_host[i]<<" ";
    }

	std::cout<<std::endl;

    hipFree(tab_device);
    
}