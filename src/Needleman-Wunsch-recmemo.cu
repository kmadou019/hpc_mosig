#include "hip/hip_runtime.h"
/**
 * \file Needleman-Wunsch-recmemo.c
 * \brief recursive implementation with memoization of Needleman-Wunsch global alignment algorithm that computes the distance between two genetic sequences 
 * \version 0.1
 * \date 03/10/2022 
 * \author Jean-Louis Roch (Ensimag, Grenoble-INP - University Grenoble-Alpes) jean-louis.roch@grenoble-inp.fr
 *
 * Documentation: see Needleman-Wunsch-recmemo.h
 * Costs of basic base opertaions (SUBSTITUTION_COST, SUBSTITUTION_UNKNOWN_COST, INSERTION_COST) are
 * defined in Needleman-Wunsch-recmemo.h
 */


#include <stdio.h>  
#include <stdlib.h> 
#include <stdbool.h>
#include <string.h> /* for strchr */
#include "utils.h"
#include "Needleman-Wunsch-recmemo.h"
#include "characters_to_base.h" /* mapping from char to base */

#define THREADS_PER_BLOCK 8

/*****************************************************************************/
   
/* Context of the memoization : passed to all recursive calls */
/** \def NOT_YET_COMPUTED
 * \brief default value for memoization of minimal distance (defined as an impossible value for a distance, -1).
 */
#define NOT_YET_COMPUTED -1L 

/** \struct NW_MemoContext
 * \brief data for memoization of recursive Needleman-Wunsch algorithm 
*/
struct NW_MemoContext 
{
    char *X ; /*!< the longest genetic sequences */
    char *Y ; /*!< the shortest genetic sequences */
    size_t M; /*!< length of X */
    size_t N; /*!< length of Y,  N <= M */
    long **memo; /*!< memoization table to store memo[0..M][0..N] (including stopping conditions phi(M,j) and phi(i,N) */
} ;

struct NW_NoMemoContext {
   char *X;
   char *Y;
   size_t M;
   size_t N;
};

#define min(a, b) ( (a < b) ? (a) : (b))

__global__ void pre_compute_min(long* tab_in, long* tab_out, long N, long i, struct NW_NoMemoContext *c){

   int j =   blockDim.x * blockIdx.x + threadIdx.x;
   
   if(j < N + 1){
      if(j == N ) tab_out[j] = 2 * isBase(c->X[i]) + tab_in[j];
      
      else if (!isBase(c->X[i])) tab_out[j] = tab_in[j];

      else if (!isBase(c->Y[j])) tab_out[j] = tab_out[j + 1]; // problem
      
      else{
         long min_tmp = (isUnknownBase(c->X[i]) ? SUBSTITUTION_UNKNOWN_COST :
                     (isSameBase(c->X[i], c->Y[j]) ? 0 : SUBSTITUTION_COST)) + tab_in[j+1];

         tab_out[j] = min(INSERTION_COST + tab_in[j], min_tmp );
      } 
         
   }


}

long EditDistance_NW_GPU(char *A, size_t lengthA, char *B, size_t lengthB) {
   _init_base_match();
   
   struct NW_NoMemoContext ctx;
   if (lengthA >= lengthB) {
      ctx.X = A;
      ctx.M = lengthA;
      ctx.Y = B;
      ctx.N = lengthB;
   } else {
      ctx.X = B;
      ctx.M = lengthB;
      ctx.Y = A;
      ctx.N = lengthA;
   }
   struct NW_NoMemoContext *c = &ctx;
   
   const long M = c->M;
   const long N = c->N;
   const long size = sizeof(long) * (N+1);
   const long size_masked = sizeof(long) * (N+1+1);
   long *h_tab_in = (long*)malloc(size_masked);
   long *h_tab_out= (long*)malloc(size_masked);
   
   long *d_tab_in ;
   long *d_tab_out;

   // alloc mem on GPU
   hipMalloc( (void**)&d_tab_in, size_masked  );
   hipMalloc( (void**)&d_tab_out, size_masked );

   hipMemcpy(d_tab_in, h_tab_in, size, hipMemcpyHostToDevice);
   
   
   h_tab_in[N] = 0;
   for (long j = N - 1; j >= 0; j--) {
      h_tab_in[j] = 2 * isBase(c->Y[j]) + h_tab_in[j + 1];
   }
   
   for (long i = M - 1; i >= 0; i--) {

      pre_compute_min<<< ceil(N/THREADS_PER_BLOCK) , THREADS_PER_BLOCK>>>(d_tab_in, d_tab_out, N, i,c);

      h_tab_out[N+1] = h_tab_out[N];
      hipMemcpy(d_tab_out, h_tab_out, size, hipMemcpyHostToDevice);
      prescan<<<ceil(N/THREADS_PER_BLOCK) , THREADS_PER_BLOCK, size_masked>>>(h_tab_out,h_tab_out, N+1+1);

      //permute tab_in and tab_out

      long *temp = h_tab_in;
      h_tab_in   = h_tab_out;
      h_tab_out  = temp;


   }

   long res = h_tab_in[0]; // Normally it has to be h_tab_out[0] but there is a permutation just before

   return res;
}
