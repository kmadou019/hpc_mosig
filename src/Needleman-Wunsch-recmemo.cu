#include "hip/hip_runtime.h"
/**
 * \file Needleman-Wunsch-recmemo.c
 * \brief recursive implementation with memoization of Needleman-Wunsch global alignment algorithm that computes the distance between two genetic sequences 
 * \version 0.1
 * \date 03/10/2022 
 * \author Jean-Louis Roch (Ensimag, Grenoble-INP - University Grenoble-Alpes) jean-louis.roch@grenoble-inp.fr
 *
 * Documentation: see Needleman-Wunsch-recmemo.h
 * Costs of basic base opertaions (SUBSTITUTION_COST, SUBSTITUTION_UNKNOWN_COST, INSERTION_COST) are
 * defined in Needleman-Wunsch-recmemo.h
 */


#include "Needleman-Wunsch-recmemo.h"
#include <stdio.h>  
#include <stdlib.h> 
#include <stdbool.h>
#include <string.h> /* for strchr */
// #include <math.h>
// #include <ctype.h> /* for toupper */

#include "characters_to_base.h" /* mapping from char to base */

/*****************************************************************************/
   
/* Context of the memoization : passed to all recursive calls */
/** \def NOT_YET_COMPUTED
 * \brief default value for memoization of minimal distance (defined as an impossible value for a distance, -1).
 */
#define NOT_YET_COMPUTED -1L 

/** \struct NW_MemoContext
 * \brief data for memoization of recursive Needleman-Wunsch algorithm 
*/
struct NW_MemoContext 
{
    char *X ; /*!< the longest genetic sequences */
    char *Y ; /*!< the shortest genetic sequences */
    size_t M; /*!< length of X */
    size_t N; /*!< length of Y,  N <= M */
    long **memo; /*!< memoization table to store memo[0..M][0..N] (including stopping conditions phi(M,j) and phi(i,N) */
} ;

struct NW_NoMemoContext {
   char *X;
   char *Y;
   size_t M;
   size_t N;
};



long EditDistance_NW_It(char *A, size_t lengthA, char *B, size_t lengthB) {
   _init_base_match();
   
   struct NW_NoMemoContext ctx;
   if (lengthA >= lengthB) {
      ctx.X = A;
      ctx.M = lengthA;
      ctx.Y = B;
      ctx.N = lengthB;
   } else {
      ctx.X = B;
      ctx.M = lengthB;
      ctx.Y = A;
      ctx.N = lengthA;
   }
   struct NW_NoMemoContext *c = &ctx;
   
   const long M = c->M;
   const long N = c->N;
   long *tab = malloc(sizeof(long) * (N+1));
   long last = 0;
   
   tab[N] = 0;
   for (long j = N - 1; j >= 0; j--) {
      tab[j] = 2 * isBase(c->Y[j]) + tab[j + 1];
   }

   for (long i = M - 1; i >= 0; i--) {
      for (long j = N; j >= 0; j--) {
         long curr = tab[j];

         if (j == N) tab[j] = 2 * isBase(c->X[i]) + tab[j];

         else if (!isBase(c->X[i])) tab[j] = tab[j];

         else if (!isBase(c->Y[j])) tab[j] = tab[j + 1];
            
         else {
            long min = (isUnknownBase(c->X[i]) ? SUBSTITUTION_UNKNOWN_COST :
                          (isSameBase(c->X[i], c->Y[j]) ? 0 : SUBSTITUTION_COST)) + last; 
            
            long val = INSERTION_COST + tab[j];      
            if (val < min) min = val;

            val = INSERTION_COST + tab[j + 1];      
            if (val < min) min = val;

            tab[j] = min;
         }

         last = curr;
      }
   }

   long res = tab[0];

   free(tab);

   return res;
}
