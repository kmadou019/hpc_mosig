#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"

#define N 16
#define THREADS_PER_BLOCK 8

__global__ void prescan(long *d_out, long *d_in, long n) {
    

    extern __shared__ long temp[];  // Allocation dynamique de la mémoire partagée
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Chargement des données dans la mémoire partagée
    temp[2 * tid] = (2 * tid < n) ? d_in[2 * tid] : 0.0f;
    temp[2 * tid + 1] = (2 * tid + 1 < n) ? d_in[2 * tid + 1] : 0.0f;
    __syncthreads();

    // Phase d'upsweep (réduction)
    for (long stride = 1; stride <= n / 2; stride *= 2) {
        long index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            temp[index] = min(temp[index] , temp[index - stride]);
        }
        __syncthreads();
    }

    // Mise à zéro du dernier élément
    if (tid == 0) {
        temp[n - 1] = INFINITY;
    }
    __syncthreads();

    // Phase de downsweep (propagation)
    for (long stride = n / 2; stride > 0; stride /= 2) {
        long index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            long t = temp[index - stride];
            temp[index - stride] = temp[index];
            temp[index] = min(temp[index], t);
        }
        __syncthreads();
    }

    // Stockage du résultat dans la mémoire globale
    if (2 * tid + 1 < n) d_out[2 * tid] = temp[2 * tid + 1];
    if (2 * tid + 1 + 1 < n) d_out[2 * tid + 1] = temp[2 * tid + 1 + 1];
}

int main() {
    long h_in[N] = {8, 3, 1, 7, 14, 4, 6, 3, 9, 2, 5, 8, 1, 7, 4, -4};
    long h_out[N];

    std::cout << "Input: ";
    for (long i = 0; i < N; i++) std::cout << h_in[i] << " ";
    std::cout << std::endl;

    long *d_in, *d_out;
    hipMalloc((void**)&d_in, N * sizeof(long));
    hipMalloc((void**)&d_out, N * sizeof(long));

    hipMemcpy(d_in, h_in, N * sizeof(long), hipMemcpyHostToDevice);

    prescan<<<2, THREADS_PER_BLOCK, N * sizeof(long)>>>(d_out, d_in, N);

    hipMemcpy(h_out, d_out, N * sizeof(long), hipMemcpyDeviceToHost);

    std::cout << "Prescan Output: ";
    for (long i = 0; i < N; i++) std::cout << h_out[i] << " ";
    std::cout << std::endl;

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
