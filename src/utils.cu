#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"
#include "characters_to_base.h" /* mapping from char to base */


#define THREADS_PER_BLOCK 8

__global__ void prescan(long *d_out, long *d_in, long n) {
    extern __shared__ long temp[];  // Allocation dynamique de la mémoire partagée
    int tid = threadIdx.x;

    // Chargement des données dans la mémoire partagée
    temp[2 * tid] = (2 * tid < n) ? d_in[2 * tid] : 0;
    temp[2 * tid + 1] = (2 * tid + 1 < n) ? d_in[2 * tid + 1] : 0;
    __syncthreads();

    // Phase d'upsweep (réduction)
    for (long stride = 1; stride <= n / 2; stride *= 2) {
        long index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            temp[index] = min(temp[index], temp[index - stride]);
        }
        __syncthreads();
    }

    // Mise à zéro du dernier élément
    if (tid == 0) {
        temp[n - 1] = INFINITY;
    }
    __syncthreads();

    // Phase de downsweep (propagation)
    for (long stride = n / 2; stride > 0; stride /= 2) {
        long index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            long t = temp[index - stride];
            temp[index - stride] = temp[index];
            temp[index] = min(temp[index],t+INSERTION_COST);
       	    if (index == 1) temp[index] -= INSERTION_COST;
       	}
        __syncthreads();
    }

    // Stockage du résultat dans la mémoire globale
    if (2 * tid + 1 < n) d_out[2 * tid] = temp[2 * tid + 1];
    if (2 * tid + 1 + 1< n) d_out[2 * tid + 1] = temp[2 * tid + 1 + 1];
}

int main() {
    long N = 15;
    long *h_in  = (long*)malloc((N+1) * sizeof(long));
    long *h_out = (long*)malloc((N+1) * sizeof(long));

    long values[N] = {8, 3, 1, 7, 14, 4, -4, 3, 9, 2, 8, 1, -7, 4, 30};
    for (int i = 0; i < N; i++) {
        h_in[i] = values[i];
    }

    std::cout << "Input: ";
    for (long i = 0; i < N; i++) std::cout << h_in[i] << " ";
    std::cout << std::endl;

    long *d_in, *d_out;
    hipMalloc((void**)&d_in, (N+1)* sizeof(long));
    hipMalloc((void**)&d_out, (N+1) * sizeof(long));

    h_in[N] = h_in[N-1];
    hipMemcpy(d_in, h_in, (N) * sizeof(long), hipMemcpyHostToDevice);

    prescan<<<2, THREADS_PER_BLOCK, (N+1) * sizeof(long)>>>(d_out, d_in, N+1);

    hipMemcpy(h_out, d_out, (N) * sizeof(long), hipMemcpyDeviceToHost);

    std::cout << "Prescan Output: ";
    for (long i = 0; i < N; i++) std::cout <<h_out[i] << " ";
    std::cout << std::endl;

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
